#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#ifdef __cplusplus
extern "C"
{
	#include "../Errors/I_Cross_Errors.h"
	#include "../TP_Cross_Global.h"
}
#endif

#include "I_Cross_Cuda.hu"
#include "I_K_Cross.hu"

__global__ void TPCross_K_BytesToHex(char *_Str, size_t _StrSize, char *_Hex)
{
	size_t _id = threadIdx.x + blockIdx.x * blockDim.x;

	size_t StartIndex = _id * _BytesToHex_ChunkSize;
	size_t EndIndex = StartIndex + _BytesToHex_ChunkSize;
	EndIndex = (EndIndex > _StrSize) ? _StrSize : EndIndex;

	for (size_t i = StartIndex; i < EndIndex; i++)
	{
		_Hex[(i * 2)]		= _BytesToHex_HexChars[(_Str[i] >> 4) & 0xF];
		_Hex[(i * 2) + 1]	= _BytesToHex_HexChars[(_Str[i])	  & 0xF];
	}
}

__device__ char Hex00_FF[256][3] =
{
	"00", "01", "02", "03", "04", "05", "06", "07", "08", "09", "0a", "0b", "0c", "0d", "0e", "0f",
	"10", "11", "12", "13", "14", "15", "16", "17", "18", "19", "1a", "1b", "1c", "1d", "1e", "1f",
	"20", "21", "22", "23", "24", "25", "26", "27", "28", "29", "2a", "2b", "2c", "2d", "2e", "2f",
	"30", "31", "32", "33", "34", "35", "36", "37", "38", "39", "3a", "3b", "3c", "3d", "3e", "3f",
	"40", "41", "42", "43", "44", "45", "46", "47", "48", "49", "4a", "4b", "4c", "4d", "4e", "4f",
	"50", "51", "52", "53", "54", "55", "56", "57", "58", "59", "5a", "5b", "5c", "5d", "5e", "5f",
	"60", "61", "62", "63", "64", "65", "66", "67", "68", "69", "6a", "6b", "6c", "6d", "6e", "6f",
	"70", "71", "72", "73", "74", "75", "76", "77", "78", "79", "7a", "7b", "7c", "7d", "7e", "7f",
	"80", "81", "82", "83", "84", "85", "86", "87", "88", "89", "8a", "8b", "8c", "8d", "8e", "8f",
	"90", "91", "92", "93", "94", "95", "96", "97", "98", "99", "9a", "9b", "9c", "9d", "9e", "9f",
	"a0", "a1", "a2", "a3", "a4", "a5", "a6", "a7", "a8", "a9", "aa", "ab", "ac", "ad", "ae", "af",
	"b0", "b1", "b2", "b3", "b4", "b5", "b6", "b7", "b8", "b9", "ba", "bb", "bc", "bd", "be", "bf",
	"c0", "c1", "c2", "c3", "c4", "c5", "c6", "c7", "c8", "c9", "ca", "cb", "cc", "cd", "ce", "cf",
	"d0", "d1", "d2", "d3", "d4", "d5", "d6", "d7", "d8", "d9", "da", "db", "dc", "dd", "de", "df",
	"e0", "e1", "e2", "e3", "e4", "e5", "e6", "e7", "e8", "e9", "ea", "eb", "ec", "ed", "ee", "ef",
	"f0", "f1", "f2", "f3", "f4", "f5", "f6", "f7", "f8", "f9", "fa", "fb", "fc", "fd", "fe", "ff",
};

__device__ int TPCROSS_K_D_GetHexIndex(char *_Str, int _index)
{
	int res = -1;
	for(int i = 0; i < 256; i++)
	{
		if(Hex00_FF[i][0] == _Str[_index] && Hex00_FF[i][1] == _Str[_index + 1])
		{
			res = i;
			break;
		}
	}
	return res;
}

__global__ void TPCross_K_HexVectorize(char *_Str, size_t _StrSize, int *_Vector)
{
	// chunk
	// for hex in str
	//		find index
	//		Increment value at index
	size_t _id = threadIdx.x + blockIdx.x * blockDim.x;
	// size_t StrSize = (int)(_StrSize / 2);

	size_t StartIndex = _id * _HexVectorize_ChunkSize;
	size_t EndIndex = StartIndex + _HexVectorize_ChunkSize;
	EndIndex = (EndIndex > _StrSize) ? _StrSize : EndIndex;

	for (int i = StartIndex; i < EndIndex; i++)
	{
		int TargetIndex = TPCROSS_K_D_GetHexIndex(_Str, (i * 2));
		if(TargetIndex >= 0)
		{
			atomicAdd(&(_Vector[TargetIndex]), 1);
		}
		else{printf("ERROR:_K_HexVectorize()\n"); return;}
	}
}

__global__ void TPCross_K_HexIndex(char *_Str, size_t _StrSize, int *_IndexTable, int *_IndexTable_ActiveIndex, int *_IndexTable_StartIndex)
{
	size_t _id = threadIdx.x + blockIdx.x * blockDim.x;

	size_t StartIndex = _id * _HexIndex_ChunkSize;
	size_t EndIndex = StartIndex + _HexIndex_ChunkSize;
	EndIndex = (EndIndex > _StrSize) ? _StrSize : EndIndex;

	for (int i = StartIndex; i < EndIndex; i++)
	{
		int TargetIndex = TPCROSS_K_D_GetHexIndex(_Str, (i * 2));
		_IndexTable[_IndexTable_StartIndex[TargetIndex] + _IndexTable_ActiveIndex[TargetIndex]] = i;
		atomicAdd(&(_IndexTable_ActiveIndex[TargetIndex]), 1);
	}
}

__device__ int TPCross_K_D_FindSubstring(
	char *_String, size_t _String_size,
	char *_SubString, size_t _SubString_size, size_t _SubString_start,
	int *_String_index_table, int *_String_start_offset, int *_String_vector)
{
	int activeHex = TPCROSS_K_D_GetHexIndex(_SubString, _SubString_start);

	int ToRet = -1;
	for (int i = 0; i < _String_vector[activeHex]; i++)
	{
		int activePin = _String_index_table[_String_start_offset[activeHex] + i];
		int found = 1;
		for (int j = 0; j < _SubString_size; j++)
		{
			int _j = (j * 2);
			if(_SubString[_SubString_start + _j] != _String[activePin + _j] || _SubString[_SubString_start + (_j + 1)] != _String[activePin + _j + 1])
			{
				found = 0;
			}
		}

		if(found == 1)
		{
			ToRet = activePin;
			break;
		}
	}
	return ToRet;
}

__global__ void TPCross_K(
	char *_source, size_t _source_size, 
	char *_target, size_t _target_size, 
	int *_source_index_table, int *_source_start_offset, int *_source_vector,
	TP_CROSS_ReferenceObj *_result, int *resultSize)
{
	size_t _id = threadIdx.x + blockIdx.x * blockDim.x;

	size_t StartIndex = _id * _Cross_ChunkSize;
	size_t EndIndex = StartIndex + _Cross_ChunkSize;
	EndIndex = (EndIndex > _target_size) ? _target_size : EndIndex;

	size_t _res_index = _Cross_Res_Chunk * _id;
	size_t _res_startIndex = 0;
	size_t _res_endIndex = _Cross_Min_Reference;
	_res_endIndex = (_res_endIndex >= (EndIndex - StartIndex)) ? (EndIndex - StartIndex) : _Cross_Min_Reference;

	int queryCache = -1;
	int _found = TP_FALSE;
	for (int i = StartIndex; i < EndIndex;)
	{
		//printf("s%d", i); printf("l%d\n", _res_endIndex);
		int query = TPCross_K_D_FindSubstring(_source, _source_size, _target, _res_endIndex, i * 2, _source_index_table, _source_start_offset, _source_vector);
		//printf("qrr: %d\n", query);

		if(query == -1 || i > EndIndex - 1 || i + _res_endIndex > EndIndex || _res_endIndex > EndIndex)
		{
			if(_found == TP_TRUE)
			{
				if(_result[_res_index].isReference == 0 && _result[_res_index].endIndex > 0){ _res_index++; }
				_result[_res_index].startIndex = queryCache;
				_result[_res_index].endIndex = _res_endIndex - 1;
				_result[_res_index].isReference = 1;

				i += _res_endIndex - 1;
				
				if(i + _Cross_Min_Reference > EndIndex)
				{
					_res_endIndex = EndIndex - i; //printf("hhh: %d\n", EndIndex - i);
				}
				else { _res_endIndex = _Cross_Min_Reference; }

				_res_index++;
			}
			else
			{
				if(_result[_res_index].startIndex == 0 && _result[_res_index].endIndex == 0){ _result[_res_index].startIndex = i; }
				_result[_res_index].endIndex += 1;
				_result[_res_index].isReference = 0;
				i++;

				if(i + _Cross_Min_Reference > EndIndex)
				{
					_res_endIndex = EndIndex - i;
				}
				else { _res_endIndex = _Cross_Min_Reference; }
				if(i >= EndIndex || i + _res_endIndex > EndIndex) { _res_index++; }
			}
			_found = TP_FALSE;
		}
		else
		{
			_found = TP_TRUE;
			queryCache = query;
			_res_endIndex++;
		}
	}

	atomicAdd(resultSize, _res_index - (_Cross_Res_Chunk * _id)); // Total result size from this thread.
}


/* if(query != -1 && i != EndIndex && _res_endIndex <= EndIndex)
{
	_res_startIndex = query;
	_res_endIndex++;
	_found = TP_TRUE;
}
else
{
	if(_found == TP_FALSE)
	{
		_result[_res_index].endIndex += 1;
		i++;
	}
	else
	{
		if(_result[_res_index].isReference == 0 && _result[_res_index].endIndex > 0)
		{
			_res_index++;
		}
		_result[_res_index].endIndex = _res_endIndex;
		_result[_res_index].startIndex = i;
		_result[_res_index].isReference = 1;
		_res_index++;
		_found = TP_FALSE;
		i += _res_endIndex;
	}
} */
