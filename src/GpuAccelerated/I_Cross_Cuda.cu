#include <hip/hip_runtime.h>

#include "I_Cross_Cuda.hu"
#include "I_K_Cross.hu"
#include "Cross_Cuda_Utils.hu"

#ifdef __cplusplus
extern "C"
{
	#include "../Errors/I_Cross_Errors.h"
	#include "../TP_Cross_Global.h"
}
#endif

char *TPCross_BytesToHex(char *_Str, size_t _StrSize)
{
	int ThreadsNeeded	= 0;
	int BlocksNeeded	= 0;
	Array1D_Cuda_KernelMap(&ThreadsNeeded, &BlocksNeeded, _StrSize, _BytesToHex_ChunkSize);

	char *_Device_Str = TPCUDA_Alloc_Str(_Str, _StrSize);
	char *_Device_Res = TPCUDA_Alloc_StrEmpty(_StrSize * 2);

	TPCross_K_BytesToHex<<<BlocksNeeded, ThreadsNeeded>>>(_Device_Str, _StrSize, _Device_Res);
	char *_Host_Res = TPCUDA_GetStr(_Device_Res, _StrSize * 2);
	_Host_Res[_StrSize * 2] = '\0';

	TPCUDA_Free_Str(_Device_Str); _Device_Str = NULL;
	TPCUDA_Free_Str(_Device_Res); _Device_Res = NULL;

	return _Host_Res;
}

int *TPCross_HexVectorize(char *_Str, size_t _StrSize)
{
	size_t StrSize = _StrSize / 2;

	int ThreadsNeeded	= 0;
	int BlocksNeeded	= 0;
	Array1D_Cuda_KernelMap(&ThreadsNeeded, &BlocksNeeded, StrSize, _BytesToHex_ChunkSize);

	char *_Device_Str = TPCUDA_Alloc_Str(_Str, _StrSize + 1);
	int *_Device_Res = TPCUDA_Alloc_IntArrayEmpty((unsigned long)256);

	TPCross_K_HexVectorize<<<BlocksNeeded, ThreadsNeeded>>>(_Device_Str, StrSize, _Device_Res);

	int *_Host_Res = TPCUDA_GetIntArray(_Device_Res, 256);

	TPCUDA_Free_Str(_Device_Str); _Device_Str = NULL;
	TPCUDA_Free_IntArray(_Device_Res); _Device_Res = NULL;
	return _Host_Res;
}

/*
 * If {_RETURN_CUDA_2DARRAY} is set to 1, then a cuda int** will be returned.
 * Else, a C ready int** will be returned.
*/
int *TPCross_HexIndex(char *_Str, size_t _StrSize, int *_Vector, int *_StartOffset, int _RETURN_CUDA_2DARRAY)
{
	// Technically, I could replace HexVectorize with this function and a reference pass within this function.
	// It would add some speed and remove some overhead. I already wrote tests for the function prior and I already
	// wrote some code using both. It makes the code more readable and it is slightly more ram efficient
	// in the case the file never reaches the sourcing phase, so I will choose to not join them.
	// Unless profiling proves otherwhise.
	size_t StrSize = _StrSize / 2;

	int ThreadsNeeded	= 0;
	int BlocksNeeded	= 0;
	Array1D_Cuda_KernelMap(&ThreadsNeeded, &BlocksNeeded, StrSize, _HexIndex_ChunkSize);

	char *_Device_Str = TPCUDA_Alloc_Str(_Str, _StrSize + 1);
	int *_Device_Res = TPCUDA_Alloc_IntArrayEmpty((size_t)StrSize);
	int *_Device_Index = TPCUDA_Alloc_IntArrayEmpty(256);

	int *_Device_StartOffset = TPCUDA_Alloc_IntArrayEmpty(256);
	hipMemcpy(_Device_StartOffset, _StartOffset, sizeof(int) * 256, hipMemcpyHostToDevice);

	TPCross_K_HexIndex<<<BlocksNeeded, ThreadsNeeded>>>(_Device_Str, StrSize, _Device_Res, _Device_Index, _Device_StartOffset);

	if(_RETURN_CUDA_2DARRAY == 0)
	{
		int *_host_Res = (int*)malloc(sizeof(int) * StrSize);
		hipMemcpy(_host_Res, _Device_Res, sizeof(int) * StrSize, hipMemcpyDeviceToHost);

		TPCUDA_Free_Str(_Device_Str); _Device_Str = NULL;
		TPCUDA_Free_IntArray(_Device_Index); _Device_Index = NULL;
		TPCUDA_Free_IntArray(_Device_Res); _Device_Res = NULL;
		TPCUDA_Free_IntArray(_Device_StartOffset); _Device_StartOffset = NULL;
		return _host_Res;
	}
	else
	{
		TPCUDA_Free_Str(_Device_Str); _Device_Str = NULL;
		TPCUDA_Free_IntArray(_Device_Index); _Device_Index = NULL;
		TPCUDA_Free_IntArray(_Device_StartOffset); _Device_StartOffset = NULL;
		return _Device_Res;
	}
}

TP_CROSS_ReferenceObj *TPCross_Cross
(char *_Source, size_t _SourceSize, char *_Target, size_t _TargetSize, int *_SourceIndexTable, int *_SourceStartOffset, int *_SourceVector, int *_returnSize, int *_retThreadCount)
{
	int ThreadsNeeded = 0;
	int BlocksNeeded = 0;
	Array1D_Cuda_KernelMap(&ThreadsNeeded, &BlocksNeeded, (_TargetSize / 2), _Cross_ChunkSize);
	(*_retThreadCount) = ThreadsNeeded;

	char *d_Source = TPCUDA_Alloc_Str(_Source, _SourceSize);
	char *d_Target = TPCUDA_Alloc_Str(_Target, _TargetSize);

	int  *d_sourceIndexTable = TPCUDA_Alloc_IntArrayEmpty(_SourceSize);
	hipMemcpy(d_sourceIndexTable, _SourceIndexTable, sizeof(int) * _SourceSize, hipMemcpyHostToDevice);
	
	int  *d_sourceStartOffset = TPCUDA_Alloc_IntArrayEmpty(256);
	hipMemcpy(d_sourceStartOffset, _SourceStartOffset, sizeof(int) * 256, hipMemcpyHostToDevice);

	int  *d_sourceVector = TPCUDA_Alloc_IntArrayEmpty(256);
	hipMemcpy(d_sourceVector, _SourceVector, sizeof(int) * 256, hipMemcpyHostToDevice);

	TP_CROSS_ReferenceObj *d_result;
	hipMalloc(&d_result, sizeof(TP_CROSS_ReferenceObj) * (_Cross_Res_Chunk * ThreadsNeeded));
	hipMemset(d_result, 0, sizeof(TP_CROSS_ReferenceObj) * (_Cross_Res_Chunk * ThreadsNeeded));

	int *d_retSize;
	hipMalloc(&d_retSize, sizeof(int));

	TPCross_K<<<BlocksNeeded, ThreadsNeeded>>>
		(d_Source, _SourceSize/2, d_Target, _TargetSize/2, d_sourceIndexTable, d_sourceStartOffset, d_sourceVector, d_result, d_retSize);

	TP_CROSS_ReferenceObj *h_results = (TP_CROSS_ReferenceObj*)malloc(sizeof(TP_CROSS_ReferenceObj) * (_Cross_Res_Chunk * ThreadsNeeded));
	hipMemcpy(h_results, d_result, sizeof(TP_CROSS_ReferenceObj) * (_Cross_Res_Chunk * ThreadsNeeded), hipMemcpyDeviceToHost);

	int *retSize = (int*)malloc(sizeof(int));
	hipMemcpy(retSize, d_retSize, sizeof(int), hipMemcpyDeviceToHost);
	(*_returnSize) = *retSize;

	free(retSize); retSize = NULL;
	hipFree(d_retSize);
	TPCUDA_Free_Str(d_Source);
	TPCUDA_Free_Str(d_Target);
	TPCUDA_Free_IntArray(d_sourceIndexTable);
	TPCUDA_Free_IntArray(d_sourceStartOffset);
	TPCUDA_Free_IntArray(d_sourceVector);
	hipFree(d_result); d_result = NULL;

	return h_results;
}

void I_TPCUDA_Free_IntArray(int *_IntArr)
{
	TPCUDA_Free_IntArray(_IntArr);
}
