#include <hip/hip_runtime.h>

#include "../TP_Cross_Global.h"
#include "I_Cross_Cuda.hu"
#include "I_K_Cross.hu"
#include "Cross_Cuda_Utils.hu"

char *TPCross_BytesToHex(char *_Str, size_t _StrSize)
{
	int ThreadsNeeded	= 0;
	int BlocksNeeded	= 0;
	Array1D_Cuda_KernelMap(&ThreadsNeeded, &BlocksNeeded, _StrSize);

	char *_Device_Str = TPCUDA_Alloc_Str(_Str, _StrSize);
	char *_Device_Res = TPCUDA_Alloc_StrEmpty(_StrSize * 2);

	TPCross_K_BytesToHex<<<BlocksNeeded, ThreadsNeeded>>>(_Device_Str, _StrSize, _Device_Res);
	char *_Host_Res = TPCUDA_GetStr(_Device_Res, _StrSize * 2);
	_Host_Res[_StrSize * 2] = '\0';

	TPCUDA_Free_Str(_Device_Str); _Device_Str = NULL;
	TPCUDA_Free_Str(_Device_Res); _Device_Res = NULL;

	return _Host_Res;
}

int *TPCross_HexVectorize(char *_Str, size_t _StrSize)
{
	size_t StrSize = _StrSize / 2;

	int ThreadsNeeded	= 0;
	int BlocksNeeded	= 0;
	Array1D_Cuda_KernelMap(&ThreadsNeeded, &BlocksNeeded, StrSize);

	char *_Device_Str = TPCUDA_Alloc_Str(_Str, _StrSize + 1);
	int *_Device_Res = TPCUDA_Alloc_IntArrayEmpty((unsigned long)256);

	TPCross_K_HexVectorize<<<BlocksNeeded, ThreadsNeeded>>>(_Device_Str, StrSize, _Device_Res);

	int *_Host_Res = TPCUDA_GetIntArray(_Device_Res, 256);

	TPCUDA_Free_Str(_Device_Str); _Device_Str = NULL;
	TPCUDA_Free_IntArray(_Device_Res); _Device_Res = NULL;
	return _Host_Res;
}

/*
 * If {_RETURN_CUDA_2DARRAY} is set to 1, then a cuda int** will be returned.
 * Else, a C ready int** will be returned.
*/
int **TPCross_HexIndex(char *_Str, size_t _StrSize, int *_Vector, int _RETURN_CUDA_2DARRAY)
{
	// Technically, I could replace HexVectorize with this function and a reference pass within this function.
	// It would add some speed and remove some overhead. I already wrote tests for the function prior and I already
	// wrote some code using both. It makes the code more readable and it is slightly more ram efficient
	// in the case the file never reaches the sourcing phase, so I will choose to not join them.
	// Unless profiling proves otherwhise.
	size_t StrSize = _StrSize / 2;

	int ThreadsNeeded	= 0;
	int BlocksNeeded	= 0;
	Array1D_Cuda_KernelMap(&ThreadsNeeded, &BlocksNeeded, StrSize);

	char *_Device_Str = TPCUDA_Alloc_Str(_Str, _StrSize + 1);
	int **_Device_Res = TPCUDA_Alloc_2DIntArrayEmpty((size_t)256);
	for (int i = 0; i < 256; i++)
	{
		_Device_Res[i] = TPCUDA_Alloc_IntArrayEmpty(_Vector[i]);
	}
	int *_Device_Index = TPCUDA_Alloc_IntArrayEmpty(256);

	TPCross_K_HexIndex<<<BlocksNeeded, ThreadsNeeded>>>(_Device_Str, StrSize, _Device_Res, _Device_Index);

	if(_RETURN_CUDA_2DARRAY == 0)
	{
		int **_host_Res = (int**)malloc(sizeof(int*) * 256);
		for (int i = 0; i < 256; i++)
		{
			_host_Res[i] = TPCUDA_GetIntArray(_Device_Res[i], _Vector[i]);
		}
		TPCUDA_Free_Str(_Device_Str); _Device_Str = NULL;
		TPCUDA_Free_IntArray(_Device_Index); _Device_Res = NULL;
		TPCUDA_Free_2DIntArray(_Device_Res, 256);
		return _host_Res;
	}
	else
	{
		TPCUDA_Free_Str(_Device_Str); _Device_Str = NULL;
		TPCUDA_Free_IntArray(_Device_Index); _Device_Res = NULL;
		return _Device_Res;
	}
}

TP_CROSS_ReferenceObj *TPCross_Cross(char *_Source, size_t _SourceSize, char *_Target, size_t _TargetSize, int **_SourceIndexTable, int *_SourceVector)
{
	char *_Device_Source			= TPCUDA_Alloc_Str(_Source, _SourceSize);
	char *_Device_Target			= TPCUDA_Alloc_Str(_Target, _TargetSize);
	int **_Device_SourceIndexTable	= _SourceIndexTable;
	int  *_Device_SourceVector		= TPCUDA_Alloc_IntArrayEmpty(256);
	hipMemcpy(_Device_SourceVector, _SourceVector, sizeof(int) * 256, hipMemcpyHostToDevice);
	
}
