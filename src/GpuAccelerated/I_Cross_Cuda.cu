#include <hip/hip_runtime.h>

#include "I_Cross_Cuda.hu"
#include "I_K_Cross.hu"
#include "Cross_Cuda_Utils.hu"

char *TPCross_BytesToHex(char *_Str, size_t _StrSize)
{
	int ThreadsNeeded	= 0;
	int BlocksNeeded	= 0;
	Array1D_Cuda_KernelMap(&ThreadsNeeded, &BlocksNeeded, _StrSize);

	char *_Device_Str = TPCUDA_Alloc_Str(_Str, _StrSize);
	char *_Device_Res = TPCUDA_Alloc_StrEmpty(_StrSize * 2);

	TPCross_K_BytesToHex<<<BlocksNeeded, ThreadsNeeded>>>(_Device_Str, _StrSize, _Device_Res);
	char *_Host_Res = TPCUDA_GetStr(_Device_Res, _StrSize * 2);
	_Host_Res[_StrSize * 2] = '\0';

	TPCUDA_Free_Str(_Device_Str); _Device_Str = NULL;
	TPCUDA_Free_Str(_Device_Res); _Device_Res = NULL;

	return _Host_Res;
}

int *TPCross_HexVectorize(char *_Str, size_t _StrSize)
{
	size_t StrSize = _StrSize / 2;

	int ThreadsNeeded	= 0;
	int BlocksNeeded	= 0;
	Array1D_Cuda_KernelMap(&ThreadsNeeded, &BlocksNeeded, StrSize);

	char *_Device_Str = TPCUDA_Alloc_Str(_Str, _StrSize + 1);
	int *_Device_Res = TPCUDA_Alloc_IntArrayEmpty((unsigned long)256);

	TPCross_K_HexVectorize<<<BlocksNeeded, ThreadsNeeded>>>(_Device_Str, StrSize, _Device_Res);

	int *_Host_Res = TPCUDA_GetIntArray(_Device_Res, 256);

	TPCUDA_Free_Str(_Device_Str); _Device_Str = NULL;
	TPCUDA_Free_IntArray(_Device_Res); _Device_Res = NULL;
	return _Host_Res;
}

