#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "I_Cross_Cuda.hu"
#include "Cross_Cuda_Utils.hu"

void Array1D_Cuda_KernelMap(int *_ThreadsNeeded, int *_BlocksNeeded, size_t _StrSize)
{
	int ThreadsNeeded = (int)ceil((double)_StrSize/_BytesToHex_ChunkSize);
	ThreadsNeeded = IfZero(ThreadsNeeded, 1);

	int BlocksNeeded = (int)ceil((double)ThreadsNeeded/_ThreadsPerBlock);
	BlocksNeeded = IfZero(BlocksNeeded, 1);

	ThreadsNeeded = (ThreadsNeeded > _ThreadsPerBlock) ? _ThreadsPerBlock : ThreadsNeeded;

	*(_ThreadsNeeded) = ThreadsNeeded;
	*(_BlocksNeeded) = BlocksNeeded;
}

char *TPCUDA_Alloc_Str(char *_Str, size_t _StrSize)
{
	char *ToRet = NULL;
	size_t ToRetSize = (sizeof(char) * (_StrSize + 1));

	hipError_t _malloc_ERR = hipMalloc(&ToRet, ToRetSize);
	if(ToRet == NULL)
	{TPCUDA_Free_Str(ToRet); return NULL; }

	hipError_t _memcpy_ERR = hipMemcpy(ToRet, _Str, ToRetSize, hipMemcpyHostToDevice);
	if(ToRet == NULL)
	{TPCUDA_Free_Str(ToRet); return NULL; }

	// printf("_malloc_ERR: %s\n", hipGetErrorString(_malloc_ERR));
	// printf("_memcpy_ERR: %s\n", hipGetErrorString(_memcpy_ERR));

	return ToRet;
}

char *TPCUDA_Alloc_StrEmpty(size_t _StrSize)
{
	char *ToRet;
	size_t ToRetSize = (sizeof(char) * (_StrSize + 1));

	hipMalloc(&ToRet, ToRetSize);
	hipMemset(ToRet, ' ', ToRetSize);

	return ToRet;
}

char *TPCUDA_GetStr(char *_Str, size_t _StrSize)
{
	size_t ToRetSize = (sizeof(char) * (_StrSize + 1));
	char *ToRet = (char*)malloc(ToRetSize);

	hipMemcpy(ToRet, _Str, ToRetSize, hipMemcpyDeviceToHost);

	return ToRet;
}

// int *TPCUDA_Alloc_IntArray()

int *TPCUDA_Alloc_IntArrayEmpty(size_t _ElementsCount)
{
	int *ToRet;
	size_t ToRetSize = (sizeof(int) * _ElementsCount);

	hipMalloc(&ToRet, ToRetSize);
	hipMemset(ToRet, 0, ToRetSize);

	return ToRet;
}

int *TPCUDA_GetIntArray(int *_IntArray, size_t _ElementsCount)
{
	size_t ToRetSize = (sizeof(int) * _ElementsCount);
	int *ToRet = (int*)malloc(ToRetSize);

	hipMemcpy(ToRet, _IntArray, ToRetSize, hipMemcpyDeviceToHost);

	return ToRet;
}

void TPCUDA_Free_Str(char *_Str)
{
	hipFree(_Str);
}

void TPCUDA_Free_IntArray(int *_IntArr)
{
	hipFree(_IntArr);
}
